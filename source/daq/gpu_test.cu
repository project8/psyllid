#include "hip/hip_runtime.h"

/*
 * gpu_test.cu
 *
 *  Created on: Nov 10, 2020
 *      Author: F. Thomas
 */

#include "gpu_test.cuh"
#include <stdio.h>


__global__ void hello_gpu()
{
	int idx = blockDim.x*blockIdx.x+threadIdx.x;

	printf("Hello from GPU thread %d\n", idx);
}

namespace psyllid{

	REGISTER_NODE_AND_BUILDER( gpu_test, "gpu-test", gpu_test_binding );

	LOGGER( plog, "gpu_test" );

	gpu_test::gpu_test() :
		f_time_length( 10 ),
		f_freq_length( 10 )
	{}

	gpu_test::~gpu_test() {}

    void gpu_test::initialize()
    {
        out_buffer< 0 >().initialize( f_time_length );
        out_buffer< 1 >().initialize( f_freq_length );
        return;
    }

    void gpu_test::execute( midge::diptera* a_midge)
    {
    	out_stream< 0 >() = in_stream< 0 >();
    	out_stream< 1 >() = in_stream< 1 >();
    	LDEBUG( plog, "Executing gpu_test" );
    	hello_gpu<<<1, 4>>>();
    	hipDeviceSynchronize();

    }

    void gpu_test::finalize()
    {
        //out_buffer< 0 >().finalize();
        out_buffer< 1 >().finalize();
        return;
    }

    gpu_test_binding::gpu_test_binding() :
            _node_binding< gpu_test, gpu_test_binding >()
    {
    }

    gpu_test_binding::~gpu_test_binding()
    {
    }

    void gpu_test_binding::do_apply_config( gpu_test* a_node, const scarab::param_node& a_config ) const
    {
        LDEBUG( plog, "Configuring gpu_test with:\n" << a_config );
        /*a_node->set_time_length( a_config.get_value( "time-length", a_node->get_time_length() ) );
        a_node->set_freq_length( a_config.get_value( "freq-length", a_node->get_freq_length() ) );
        a_node->set_udp_buffer_size( a_config.get_value( "udp-buffer-size", a_node->get_udp_buffer_size() ) );
        a_node->set_time_sync_tol( a_config.get_value( "time-sync-tol", a_node->get_time_sync_tol() ) );
        a_node->set_start_paused( a_config.get_value( "start-paused", a_node->get_start_paused() ) );
        a_node->set_force_time_first( a_config.get_value( "force-time-first", a_node->get_force_time_first() ) );*/
        return;
    }

    void gpu_test_binding::do_dump_config( const gpu_test* a_node, scarab::param_node& a_config ) const
    {
        LDEBUG( plog, "Dumping gpu_test configuration" );
        /*a_config.add( "time-length", scarab::param_value( a_node->get_time_length() ) );
        a_config.add( "freq-length", scarab::param_value( a_node->get_freq_length() ) );
        a_config.add( "udp-buffer-size", scarab::param_value( a_node->get_udp_buffer_size() ) );
        a_config.add( "time-sync-tol", scarab::param_value( a_node->get_time_sync_tol() ) );
        a_config.add( "start-paused", scarab::param_value( a_node->get_start_paused() ) );
        a_config.add( "force-time-first", scarab::param_value( a_node->get_force_time_first() ) );*/
        return;

    }

    bool gpu_test_binding::do_run_command( gpu_test* a_node, const std::string& a_cmd, const scarab::param_node& ) const
    {
        /*if( a_cmd == "freq-only" )
        {
            a_node->switch_to_freq_only();
            return true;
        }
        else if( a_cmd == "time-and-freq" )
        {
            a_node->switch_to_time_and_freq();
            return true;
        }
        else
        {
            LWARN( plog, "Unrecognized command: <" << a_cmd << ">" );
            return false;
        }*/

    	LDEBUG( plog, "Do run command gpu_test" );
    	return true;
    }
}

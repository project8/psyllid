#include "hip/hip_runtime.h"

/*
 * gpu_test.cu
 *
 *  Created on: Nov 10, 2020
 *      Author: F. Thomas
 */

#include "gpu_test.cuh"
#include <stdio.h>


__global__ void hello_gpu()
{
	int idx = blockDim.x*blockIdx.x+threadIdx.x;

	printf("Hello from GPU thread %d\n", idx);
}

namespace psyllid{

	REGISTER_NODE_AND_BUILDER( gpu_test, "gpu-test", gpu_test_binding );

	LOGGER( plog, "gpu_test" );

	gpu_test::gpu_test() :
		f_time_length( 10 ),
		f_freq_length( 10 )
	{}

	gpu_test::~gpu_test() {}

    void gpu_test::initialize()
    {
        out_buffer< 0 >().initialize( f_time_length );
        out_buffer< 1 >().initialize( f_freq_length );
        return;
    }

    void gpu_test::execute( midge::diptera* a_midge)
    {
    	//out_stream< 0 >() = in_stream< 0 >();
    	//out_stream< 1 >() = in_stream< 1 >();
    	LDEBUG( plog, "Executing gpu_test" );
    	hello_gpu<<<1, 4>>>();
    	hipDeviceSynchronize();

    }

    void gpu_test::finalize()
    {
        //out_buffer< 0 >().finalize();
        out_buffer< 1 >().finalize();
        return;
    }

    gpu_test_binding::gpu_test_binding() :
            _node_binding< gpu_test, gpu_test_binding >()
    {
    }

    gpu_test_binding::~gpu_test_binding()
    {
    }

    void gpu_test_binding::do_apply_config( gpu_test* a_node, const scarab::param_node& a_config ) const
    {
        LDEBUG( plog, "Configuring gpu_test with:\n" << a_config );
        return;
    }

    void gpu_test_binding::do_dump_config( const gpu_test* a_node, scarab::param_node& a_config ) const
    {
        LDEBUG( plog, "Dumping gpu_test configuration" );
        return;

    }

    bool gpu_test_binding::do_run_command( gpu_test* a_node, const std::string& a_cmd, const scarab::param_node& ) const
    {

    	LDEBUG( plog, "Do run command gpu_test" );
    	return true;
    }
}
